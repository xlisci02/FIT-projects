#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xlisci02
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate gravitation velocity
 * @param p_in       - input particles data
 * @param p_out      - output particles data
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{

  // Get global tread id.
  int p1_index = threadIdx.x + blockIdx.x * blockDim.x;

  // If thread index within range
  if (p1_index < N){
    float r, dx, dy, dz;

    // Preloading of particle data to registers
    float pos_x = p_in.pos_x[p1_index];
    float pos_y = p_in.pos_y[p1_index];
    float pos_z = p_in.pos_z[p1_index];
    float weight = p_in.weight[p1_index];
    float vel_x = p_in.vel_x[p1_index];
    float vel_y = p_in.vel_y[p1_index];
    float vel_z = p_in.vel_z[p1_index];

    // Temporary variables for partial sums
    float tmp_vel_x = 0.0f, tmp_vel_y = 0.0f, tmp_vel_z = 0.0f;

     // Computing gravitation & collision velocity
    for(int p2_index = 0; p2_index < N; p2_index++){
      // Preload to register
      float w = p_in.weight[p2_index];
      // Precompute value to registers
      float Fg_dt_m2_r = -G * dt * w;
      // Get the position differences
      dx = pos_x - p_in.pos_x[p2_index];
      dy = pos_y - p_in.pos_y[p2_index];
      dz = pos_z - p_in.pos_z[p2_index];

      r = sqrt(dx*dx + dy*dy + dz*dz);

      // Gravitation
      Fg_dt_m2_r /= (r * r * r + FLT_MIN);

      tmp_vel_x += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dx : 0.0f;
      tmp_vel_y += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dy : 0.0f;
      tmp_vel_z += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dz : 0.0f;

      // Collision
      if (r > 0.0f && r < COLLISION_DISTANCE){
        // Precompute values to registers
        float weight_sum = weight + w;
        float weight_diff = weight - w;
        float weight_double = w * 2;

        tmp_vel_x += ((vel_x * (weight_diff) + weight_double* p_in.vel_x[p2_index]) / (weight_sum)) - vel_x;
        tmp_vel_y += ((vel_y * (weight_diff) + weight_double* p_in.vel_y[p2_index]) / (weight_sum)) - vel_y;
        tmp_vel_z += ((vel_z * (weight_diff) + weight_double* p_in.vel_z[p2_index]) / (weight_sum)) - vel_z;
      }
    }
    // Update velocities and positions
    p_out.vel_x[p1_index] = vel_x + tmp_vel_x;
    p_out.vel_y[p1_index] = vel_y + tmp_vel_y;
    p_out.vel_z[p1_index] = vel_z + tmp_vel_z;

    p_out.pos_x[p1_index] = pos_x + ((vel_x + tmp_vel_x) * dt);
    p_out.pos_y[p1_index] = pos_y + ((vel_y + tmp_vel_y) * dt);
    p_out.pos_z[p1_index] = pos_z + ((vel_z + tmp_vel_z) * dt);
  }
}// end of calculate_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting points and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
