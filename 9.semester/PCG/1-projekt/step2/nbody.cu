#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xlisci02
 */

 #include <cmath>
 #include <cfloat>
 #include "nbody.h"

 /**
  * CUDA kernel to calculate gravitation velocity
  * @param p_in       - input particles data
  * @param p_out      - output particles data
  * @param N       - Number of particles
  * @param dt      - Size of the time step
  */
 __global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
 {
  extern __shared__ float shared_particles_data[];

   // Get global tread id.
  int p1_index = threadIdx.x + blockIdx.x * blockDim.x;

  // If thread index within range
  if (p1_index < N){
    float r, dx, dy, dz;

    // Preloading of particle data to registers
    float pos_x = p_in.pos_x[p1_index];
    float pos_y = p_in.pos_y[p1_index];
    float pos_z = p_in.pos_z[p1_index];
    float weight = p_in.weight[p1_index];
    float vel_x = p_in.vel_x[p1_index];
    float vel_y = p_in.vel_y[p1_index];
    float vel_z = p_in.vel_z[p1_index];

    // Temporary variables for partial sums
    float tmp_vel_x = 0.0f, tmp_vel_y = 0.0f, tmp_vel_z = 0.0f;

    // Save pointers to shared memory
    float *sh_pos_x = &shared_particles_data[0];
    float *sh_pos_y = &shared_particles_data[blockDim.x];
    float *sh_pos_z = &shared_particles_data[2 * blockDim.x];
    float *sh_weight = &shared_particles_data[3 * blockDim.x];
    float *sh_vel_x = &shared_particles_data[4 * blockDim.x];
    float *sh_vel_y = &shared_particles_data[5 * blockDim.x];
    float *sh_vel_z = &shared_particles_data[6 * blockDim.x];

    // Loop, moving with the size of the tile (threads/block)
    for(int i = 0, tile = 0; i < N; i += blockDim.x, tile++){

      // Index with the respect to tile
      int idx = tile * blockDim.x + threadIdx.x;

      // Load current tile data from global memory to shared memory
      sh_pos_x[threadIdx.x] =  p_in.pos_x[idx];
      sh_pos_y[threadIdx.x] =  p_in.pos_y[idx];
      sh_pos_z[threadIdx.x] =  p_in.pos_z[idx];
      sh_weight[threadIdx.x] =  p_in.weight[idx];
      sh_vel_x[threadIdx.x] =  p_in.vel_x[idx];
      sh_vel_y[threadIdx.x] =  p_in.vel_y[idx];
      sh_vel_z[threadIdx.x] =  p_in.vel_z[idx];

      // Synchronization on block level
      // Every thread should have the same data in shared memory before continuing
      __syncthreads();

      // Computing gravitation & collision velocity using shared memory
      for (int j = 0; j < blockDim.x; j++){
        float w = sh_weight[j];
        // Precompute value to registers
        float Fg_dt_m2_r = -G * dt * w;
        // Get the position differences
        dx = pos_x - sh_pos_x[j];
        dy = pos_y - sh_pos_y[j];
        dz = pos_z - sh_pos_z[j];

        r = sqrt(dx*dx + dy*dy + dz*dz);

        // Gravitation
        Fg_dt_m2_r /= (r * r * r + FLT_MIN);

        tmp_vel_x += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dx : 0.0f;
        tmp_vel_y += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dy : 0.0f;
        tmp_vel_z += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dz : 0.0f;

        // Collision
        if (r > 0.0f && r < COLLISION_DISTANCE){
          // Precompute values to registers
          float weight_sum = weight + w;
          float weight_diff = weight - w;
          float weight_double = w * 2;

          tmp_vel_x += ((vel_x * (weight_diff) + weight_double* sh_vel_x[j]) / (weight_sum)) - vel_x;
          tmp_vel_y += ((vel_y * (weight_diff) + weight_double* sh_vel_y[j]) / (weight_sum)) - vel_y;
          tmp_vel_z += ((vel_z * (weight_diff) + weight_double* sh_vel_z[j]) / (weight_sum)) - vel_z;

        }
      }
      // Thread synchronization before writing to shared memory in next iteration
      __syncthreads();
    }
    // Update velocities and positions
    p_out.vel_x[p1_index] = vel_x + tmp_vel_x;
    p_out.vel_y[p1_index] = vel_y + tmp_vel_y;
    p_out.vel_z[p1_index] = vel_z + tmp_vel_z;

    p_out.pos_x[p1_index] = pos_x + ((vel_x + tmp_vel_x) * dt);
    p_out.pos_y[p1_index] = pos_y + ((vel_y + tmp_vel_y) * dt);
    p_out.pos_z[p1_index] = pos_z + ((vel_z + tmp_vel_z) * dt);
   }
 }// end of calculate_velocity
 //----------------------------------------------------------------------------------------------------------------------

 /**
  * CPU implementation of the Center of Mass calculation
  * @param particles - All particles in the system
  * @param N         - Number of particles
  */
 __host__ float4 centerOfMassCPU(MemDesc& memDesc)
 {
   float4 com = {0 ,0, 0, 0};

   for(int i = 0; i < memDesc.getDataSize(); i++)
   {
     // Calculate the vector on the line connecting points and most recent position of center-of-mass
     const float dx = memDesc.getPosX(i) - com.x;
     const float dy = memDesc.getPosY(i) - com.y;
     const float dz = memDesc.getPosZ(i) - com.z;

     // Calculate weight ratio only if at least one particle isn't massless
     const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                           ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

     // Update position and weight of the center-of-mass according to the weight ration and vector
     com.x += dx * dw;
     com.y += dy * dw;
     com.z += dz * dw;
     com.w += memDesc.getWeight(i);
   }
   return com;
 }// enf of centerOfMassCPU
 //----------------------------------------------------------------------------------------------------------------------
