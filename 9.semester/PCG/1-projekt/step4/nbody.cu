#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xlisci02
 */

 #include <cmath>
 #include <cfloat>
 #include "nbody.h"

 /**
  * CUDA kernel to calculate gravitation velocity
  * @param p_in       - input particles data
  * @param p_out      - output particles data
  * @param N       - Number of particles
  * @param dt      - Size of the time step
  */
 __global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
 {
  extern __shared__ float shared_particles_data[];

   // Get global tread id.
  int p1_index = threadIdx.x + blockIdx.x * blockDim.x;

  if (p1_index < N){
    float r, dx, dy, dz;

    float pos_x = p_in.pos_x[p1_index];
    float pos_y = p_in.pos_y[p1_index];
    float pos_z = p_in.pos_z[p1_index];
    float weight = p_in.weight[p1_index];
    float vel_x = p_in.vel_x[p1_index];
    float vel_y = p_in.vel_y[p1_index];
    float vel_z = p_in.vel_z[p1_index];

    float tmp_vel_x = 0.0f, tmp_vel_y = 0.0f, tmp_vel_z = 0.0f;

    float *sh_pos_x = &shared_particles_data[0];
    float *sh_pos_y = &shared_particles_data[blockDim.x];
    float *sh_pos_z = &shared_particles_data[2 * blockDim.x];
    float *sh_weight = &shared_particles_data[3 * blockDim.x];
    float *sh_vel_x = &shared_particles_data[4 * blockDim.x];
    float *sh_vel_y = &shared_particles_data[5 * blockDim.x];
    float *sh_vel_z = &shared_particles_data[6 * blockDim.x];


    for(int i = 0, tile = 0; i < N; i += blockDim.x, tile++){

      int idx = tile * blockDim.x + threadIdx.x;

      sh_pos_x[threadIdx.x] =  p_in.pos_x[idx];
      sh_pos_y[threadIdx.x] =  p_in.pos_y[idx];
      sh_pos_z[threadIdx.x] =  p_in.pos_z[idx];
      sh_weight[threadIdx.x] =  p_in.weight[idx];
      sh_vel_x[threadIdx.x] =  p_in.vel_x[idx];
      sh_vel_y[threadIdx.x] =  p_in.vel_y[idx];
      sh_vel_z[threadIdx.x] =  p_in.vel_z[idx];
      __syncthreads();

      for (int j = 0; j < blockDim.x; j++){
        float w = sh_weight[j];
        float Fg_dt_m2_r = -G * dt * w;
        dx = pos_x - sh_pos_x[j];
        dy = pos_y - sh_pos_y[j];
        dz = pos_z - sh_pos_z[j];

        r = sqrt(dx*dx + dy*dy + dz*dz);

        // gravitation
        Fg_dt_m2_r /= (r * r * r + FLT_MIN);

        tmp_vel_x += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dx : 0.0f;
        tmp_vel_y += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dy : 0.0f;
        tmp_vel_z += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dz : 0.0f;

        // collision
        if (r > 0.0f && r < COLLISION_DISTANCE){
          float weight_sum = weight + w;
          float weight_diff = weight - w;
          float weight_double = w * 2;

          tmp_vel_x += ((vel_x * (weight_diff) + weight_double* sh_vel_x[j]) / (weight_sum)) - vel_x;
          tmp_vel_y += ((vel_y * (weight_diff) + weight_double* sh_vel_y[j]) / (weight_sum)) - vel_y;
          tmp_vel_z += ((vel_z * (weight_diff) + weight_double* sh_vel_z[j]) / (weight_sum)) - vel_z;

        }
      }
      __syncthreads();
    }
    // update
    p_out.vel_x[p1_index] = vel_x + tmp_vel_x;
    p_out.vel_y[p1_index] = vel_y + tmp_vel_y;
    p_out.vel_z[p1_index] = vel_z + tmp_vel_z;

    p_out.pos_x[p1_index] = pos_x + ((vel_x + tmp_vel_x) * dt);
    p_out.pos_y[p1_index] = pos_y + ((vel_y + tmp_vel_y) * dt);
    p_out.pos_z[p1_index] = pos_z + ((vel_z + tmp_vel_z) * dt);
   }
 }// end of calculate_velocity
 //----------------------------------------------------------------------------------------------------------------------


/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
 __global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
 {
  // Shared memory
  extern __shared__ float center_of_mass_data[];
  // Get pointers to shared memory.
  float * sh_com_dataX = &center_of_mass_data[0];
  float * sh_com_dataY = &center_of_mass_data[blockDim.x];
  float * sh_com_dataZ = &center_of_mass_data[2 * blockDim.x];
  float * sh_com_dataW = &center_of_mass_data[3 * blockDim.x];

  // Thread id
  unsigned int tid = threadIdx.x;

  // Zero shared memory
  float myX = 0.0f;
  float myY = 0.0f;
  float myZ = 0.0f;
  float myW = 0.0f;

  // In case that we have more particles than threads
  for (unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += gridDim.x  * blockDim.x){
    // Sum only product of position and weight,
    // but in registers, not in shared memory
    myX += p.pos_x[idx] * p.weight[idx];
    myY += p.pos_y[idx] * p.weight[idx];
    myZ += p.pos_z[idx] * p.weight[idx];
    myW += p.weight[idx];
  }

  // Warp-synchronous reduction, same as in the PCG lecture/lab
  for (unsigned int stride = 16; stride > 0; stride >>= 1)
    myX += __shfl_down_sync(0xffffffff, myX, stride);
  for (unsigned int stride = 16; stride > 0; stride >>= 1)
    myY += __shfl_down_sync(0xffffffff, myY, stride);
  for (unsigned int stride = 16; stride > 0; stride >>= 1)
    myZ += __shfl_down_sync(0xffffffff, myZ, stride);
  for (unsigned int stride = 16; stride > 0; stride >>= 1)
    myW += __shfl_down_sync(0xffffffff, myW, stride);

  // Stacking results together, prevent warp divergence
  if(tid % 32 == 0){
    sh_com_dataX[tid/32] = myX;
    sh_com_dataY[tid/32] = myY;
    sh_com_dataZ[tid/32] = myZ;
    sh_com_dataW[tid/32] = myW;
  }
  // Synchronize, all threads in a block should have same shared data
  __syncthreads();

  // Reduction using shared memory as in step3.1
  // but now we have 32 times smaller stride at the begining
  // because of warp-synchronous reduction used above
  for (unsigned int stride = blockDim.x / 64; stride > 0; stride >>= 1){
    if (tid < stride){
      unsigned int neigh_tid = tid + stride;
      sh_com_dataX[tid] += sh_com_dataX[neigh_tid];
      sh_com_dataY[tid] += sh_com_dataY[neigh_tid];
      sh_com_dataZ[tid] += sh_com_dataZ[neigh_tid];
      sh_com_dataW[tid] += sh_com_dataW[neigh_tid];
    }
    // Synchronize, before writing to shared memory in next iteration
    __syncthreads();
  }

  // Sum all partial results into global memory,
  // after sum we have to divide comX, comY, comZ by total weight (on CPU)
  if(!tid ){ // only one thread from a block writes to global memory
    // Trying to lock using atomic change and swap operation
    while(atomicCAS(lock, 0, 1));
    // Critical section
    *comX += sh_com_dataX[0];
    *comY += sh_com_dataY[0];
    *comZ += sh_com_dataZ[0];
    *comW += sh_com_dataW[0];
    // Unlock the lock using atomic exchange operation
    atomicExch(lock, 0);
  }


 }// end of centerOfMass
 //----------------------------------------------------------------------------------------------------------------------

 /**
  * CPU implementation of the Center of Mass calculation
  * @param particles - All particles in the system
  * @param N         - Number of particles
  */
 __host__ float4 centerOfMassCPU(MemDesc& memDesc)
 {
   float4 com = {0 ,0, 0, 0};

   for(int i = 0; i < memDesc.getDataSize(); i++)
   {
     // Calculate the vector on the line connecting points and most recent position of center-of-mass
     const float dx = memDesc.getPosX(i) - com.x;
     const float dy = memDesc.getPosY(i) - com.y;
     const float dz = memDesc.getPosZ(i) - com.z;

     // Calculate weight ratio only if at least one particle isn't massless
     const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                           ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

     // Update position and weight of the center-of-mass according to the weight ration and vector
     com.x += dx * dw;
     com.y += dy * dw;
     com.z += dz * dw;
     com.w += memDesc.getWeight(i);
   }
   return com;
 }// enf of centerOfMassCPU
 //----------------------------------------------------------------------------------------------------------------------
