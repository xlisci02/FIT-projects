#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xlisci02
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate gravitation velocity
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_gravitation_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{

  // Get global tread id
  int p1_index = threadIdx.x + blockIdx.x * blockDim.x;

  // If thread index within range
  if (p1_index < N){
    float r, dx, dy, dz;

    // Preloading of particle position data to registers
    float pos_x = p.pos_x[p1_index];
    float pos_y = p.pos_y[p1_index];
    float pos_z = p.pos_z[p1_index];

    // Temporary variables for partial sums
    float tmp_vel_x = 0.0f, tmp_vel_y = 0.0f, tmp_vel_z = 0.0f;

    // Computing gravitation velocity (from CPU, version 2)
    for(int p2_index = 0; p2_index < N; p2_index++){
      // First, we compute what we can
      float Fg_dt_m2_r = -G * dt * p.weight[p2_index];
      // Get the position differences
      dx = pos_x - p.pos_x[p2_index];
      dy = pos_y - p.pos_y[p2_index];
      dz = pos_z - p.pos_z[p2_index];

      // Compute r, based on the position differencies
      r = sqrt(dx*dx + dy*dy + dz*dz);
      // Finish computing Fg_dt_m2_r with r
      Fg_dt_m2_r /= (r * r * r + FLT_MIN);

      // Compute impact of particle on index p2_index
      tmp_vel_x += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dx : 0.0f;
      tmp_vel_y += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dy : 0.0f;
      tmp_vel_z += (r > COLLISION_DISTANCE) ? Fg_dt_m2_r * dz : 0.0f;
    }
    // Use temp velocity arrays to save the velocity changes
    tmp_vel.x[p1_index] = tmp_vel_x;
    tmp_vel.y[p1_index] = tmp_vel_y;
    tmp_vel.z[p1_index] = tmp_vel_z;
  }
}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to calculate collision velocity
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_collision_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  // Get global tread id.
  int p1_index = threadIdx.x + blockIdx.x * blockDim.x;

  // If thread index within range
  if (p1_index < N){
    float r, dx, dy, dz;

    // Preloading of particle data to registers
    float pos_x = p.pos_x[p1_index];
    float pos_y = p.pos_y[p1_index];
    float pos_z = p.pos_z[p1_index];
    float weight = p.weight[p1_index];
    float vel_x = p.vel_x[p1_index];
    float vel_y = p.vel_y[p1_index];
    float vel_z = p.vel_z[p1_index];

    // Temporary variables for partial sums
    float tmp_vel_x = 0.0f, tmp_vel_y = 0.0f, tmp_vel_z = 0.0f;

    // Computing gravitation velocity (from CPU, version 2)
    for(int p2_index = 0; p2_index < N; p2_index++){
      // Get the position differences
      dx = pos_x - p.pos_x[p2_index];
      dy = pos_y - p.pos_y[p2_index];
      dz = pos_z - p.pos_z[p2_index];

      // Compute r, based on the position differencies
      r = sqrtf(dx*dx + dy*dy + dz*dz);

      // if r is within range
      if (r > 0.0f && r < COLLISION_DISTANCE){
        // Precompute values to registers
        float w = p.weight[p2_index];
        float weight_sum = weight + w;
        float weight_diff = weight - w;
        float weight_double = w * 2;

        // Compute impact of particle on index p2_index
        tmp_vel_x += ((vel_x * (weight_diff) + weight_double* p.vel_x[p2_index]) / (weight_sum)) - vel_x;
        tmp_vel_y += ((vel_y * (weight_diff) + weight_double* p.vel_y[p2_index]) / (weight_sum)) - vel_y;
        tmp_vel_z += ((vel_z * (weight_diff) + weight_double* p.vel_z[p2_index]) / (weight_sum)) - vel_z;
      }
    }
    // Use temp velocity arrays to save the velocity changes
    tmp_vel.x[p1_index] += tmp_vel_x;
    tmp_vel.y[p1_index] += tmp_vel_y;
    tmp_vel.z[p1_index] += tmp_vel_z;
  }
}// end of calculate_collision_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void update_particle(t_particles p, t_velocities tmp_vel, int N, float dt)
{
  int p1_index = threadIdx.x + blockIdx.x * blockDim.x;
  if (p1_index < N){
    p.vel_x[p1_index] += tmp_vel.x[p1_index];
    p.vel_y[p1_index] += tmp_vel.y[p1_index];
    p.vel_z[p1_index] += tmp_vel.z[p1_index];

    p.pos_x[p1_index] += p.vel_x[p1_index] * dt;
    p.pos_y[p1_index] += p.vel_y[p1_index] * dt;
    p.pos_z[p1_index] += p.vel_z[p1_index] * dt;
  }
}// end of update_particle
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting points and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
