#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xlisci02
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"
#include "h5Helper.h"

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  // Time measurement
  struct timeval t1, t2;

  if (argc != 10)
  {
    printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    exit(1);
  }

  // Number of particles
  const int N           = std::stoi(argv[1]);
  // Length of time step
  const float dt        = std::stof(argv[2]);
  // Number of steps
  const int steps       = std::stoi(argv[3]);
  // Number of thread blocks
  const int thr_blc     = std::stoi(argv[4]);
  // Write frequency
  int writeFreq         = std::stoi(argv[5]);
  // number of reduction threads
  const int red_thr     = std::stoi(argv[6]);
  // Number of reduction threads/blocks
  const int red_thr_blc = std::stoi(argv[7]);

  // Size of the simulation CUDA gird - number of blocks
  const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
  // Size of the reduction CUDA grid - number of blocks
  const size_t reductionGrid  = (red_thr + red_thr_blc - 1) / red_thr_blc;

  // Log benchmark setup
  printf("N: %d\n", N);
  printf("dt: %f\n", dt);
  printf("steps: %d\n", steps);
  printf("threads/block: %d\n", thr_blc);
  printf("blocks/grid: %lu\n", simulationGrid);
  printf("reduction threads/block: %d\n", red_thr_blc);
  printf("reduction blocks/grid: %lu\n", reductionGrid);

  const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
  writeFreq = (writeFreq > 0) ?  writeFreq : 0;


  t_particles particles_cpu;
  t_com com_cpu;
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                            FILL IN: CPU side memory allocation (step 0)                                          //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  particles_cpu.pos_x = (float *) malloc(N * sizeof(float));
  particles_cpu.pos_y = (float *) malloc(N * sizeof(float));
  particles_cpu.pos_z = (float *) malloc(N * sizeof(float));
  particles_cpu.vel_x = (float *) malloc(N * sizeof(float));
  particles_cpu.vel_y = (float *) malloc(N * sizeof(float));
  particles_cpu.vel_z = (float *) malloc(N * sizeof(float));
  particles_cpu.weight = (float *) malloc(N * sizeof(float));

  // Center of mass structure allocation
  com_cpu.x = (float *) malloc(sizeof(float));
  com_cpu.y = (float *) malloc(sizeof(float));
  com_cpu.z = (float *) malloc(sizeof(float));
  com_cpu.w = (float *) malloc(sizeof(float));

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                              FILL IN: memory layout descriptor (step 0)                                          //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                      Stride of two               Offset of the first
   *  Data pointer        consecutive elements        element in floats,
   *                      in floats, not bytes        not bytes
  */
  MemDesc md(
        particles_cpu.pos_x,                1,                          0,              // Postition in X
        particles_cpu.pos_y,                1,                          0,              // Postition in Y
        particles_cpu.pos_z,                1,                          0,              // Postition in Z
        particles_cpu.vel_x,                1,                          0,              // Velocity in X
        particles_cpu.vel_y,                1,                          0,              // Velocity in Y
        particles_cpu.vel_z,                1,                          0,              // Velocity in Z
        particles_cpu.weight,               1,                          0,              // Weight
        N,                                                                  // Number of particles
        recordsNum);                                                        // Number of records in output file

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::cerr<<e.what()<<std::endl;
    return -1;
  }

  // GPU structures and variable
  t_particles p_in, p_out;
  t_com com_gpu;
  int *lock;
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                  FILL IN: GPU side memory allocation (step 0)                                    //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  hipMalloc<float>(&(p_in.pos_x), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_in.pos_y), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_in.pos_z), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_in.vel_x), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_in.vel_y), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_in.vel_z), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_in.weight), (size_t) (N * sizeof(float)));

  hipMalloc<float>(&(p_out.pos_x), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_out.pos_y), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_out.pos_z), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_out.vel_x), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_out.vel_y), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_out.vel_z), (size_t) (N * sizeof(float)));
  hipMalloc<float>(&(p_out.weight), (size_t) (N * sizeof(float)));

  // COM -- Center Of Mass allocation in global memory
  hipMalloc<float>(&(com_gpu.x), (size_t)(sizeof(float)));
  hipMalloc<float>(&(com_gpu.y), (size_t)(sizeof(float)));
  hipMalloc<float>(&(com_gpu.z), (size_t)(sizeof(float)));
  hipMalloc<float>(&(com_gpu.w), (size_t)(sizeof(float)));
  hipMalloc<int>(&(lock), (size_t)(sizeof(int)));


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                       FILL IN: memory transfers (step 0)                                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  hipMemcpy(p_in.pos_x, particles_cpu.pos_x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(p_in.pos_y, particles_cpu.pos_y, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(p_in.pos_z, particles_cpu.pos_z, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(p_in.vel_x, particles_cpu.vel_x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(p_in.vel_y, particles_cpu.vel_y, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(p_in.vel_z, particles_cpu.vel_z, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(p_in.weight, particles_cpu.weight, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(p_out.weight, particles_cpu.weight, N * sizeof(float), hipMemcpyHostToDevice);

  // Reset COM values and initialize lock
  hipMemset(com_gpu.x, 0, 1);
  hipMemset(com_gpu.y, 0, 1);
  hipMemset(com_gpu.z, 0, 1);
  hipMemset(com_gpu.w, 0, 1);
  hipMemset(lock, 0, 1);

  gettimeofday(&t1, 0);

  // Helper structure for exchange inside loop
  t_particles p_in_copy;
  for(int s = 0; s < steps; s++)
  {

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                       FILL IN: kernels invocation (step 0)                                     //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    calculate_velocity<<<simulationGrid, thr_blc, thr_blc*sizeof(float)*7>>>(p_in, p_out, N, dt);
    // particle data arrays exchange
    p_in_copy = p_in;
    p_in = p_out;
    p_out = p_in_copy;

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                          FILL IN: synchronization  (step 4)                                    //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    if (writeFreq > 0 && (s % writeFreq == 0))
    {
      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
      //                          FILL IN: synchronization and file access logic (step 4)                             //
      //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    }
  }


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                           //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  hipDeviceSynchronize();
  // calling kernel with the size of shared memory
  centerOfMass<<<simulationGrid, thr_blc, thr_blc * sizeof(float) * 5>>>(p_in, com_gpu.x, com_gpu.y, com_gpu.z, com_gpu.w, lock, N);
  gettimeofday(&t2, 0);

  // Approximate simulation wall time
  double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
  printf("Time: %f s\n", t);


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                             FILL IN: memory transfers for particle data (step 0)                                 //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  float4 comOnGPU;

  hipMemcpy(particles_cpu.pos_x, p_in.pos_x, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.pos_y, p_in.pos_y, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.pos_z, p_in.pos_z, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_x, p_in.vel_x, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_y, p_in.vel_y, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.vel_z, p_in.vel_z, N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(particles_cpu.weight, p_in.weight, N * sizeof(float), hipMemcpyDeviceToHost);


  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  // Copy computed COM from GPU on CPU
  hipMemcpy(com_cpu.x , com_gpu.x, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(com_cpu.y , com_gpu.y, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(com_cpu.z , com_gpu.z, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(com_cpu.w , com_gpu.w, sizeof(float), hipMemcpyDeviceToHost);

  // Division by total weight
  comOnGPU.x = *com_cpu.x / (*com_cpu.w);
  comOnGPU.y = *com_cpu.y / (*com_cpu.w);
  comOnGPU.z = *com_cpu.z / (*com_cpu.w);
  comOnGPU.w = *com_cpu.w;


  float4 comOnCPU = centerOfMassCPU(md);

  std::cout << "Center of mass on CPU:" << std::endl
            << comOnCPU.x <<", "
            << comOnCPU.y <<", "
            << comOnCPU.z <<", "
            << comOnCPU.w
            << std::endl;

  std::cout << "Center of mass on GPU:" << std::endl
            << comOnGPU.x<<", "
            << comOnGPU.y<<", "
            << comOnGPU.z<<", "
            << comOnGPU.w
            << std::endl;

  // Writing final values to the file
  h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
  h5Helper.writeParticleDataFinal();

  return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
